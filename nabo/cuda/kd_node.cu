#include "hip/hip_runtime.h"
//CUDA runtime for KD_nodes



/*Roadmap: 

			(We are here)
 			 |
			\ /
			 v
Core functionality -> Dynamic Parralelism Experimentation -> Linking to existing libnabo framework -> optimization -> finalization

/Optimization |= (Search key sorting, linearizing the KD tree, improving GPU caching for node heaps)/

EST: Probably a month? Unsure. I need to look through the rest of the SDK.*/
#define maximum_depth 22
#define dim_count 3
#define K_size 16
#ifndef FLOAT_MAX 
#define FLOAT_MAX 33554430.0f
#endif 
#define BLOCK_SIZE 32
#define max_rad 256
//If coordinates are within 5% of eachother when compared to their cluster maximimum, treat them as a single point. To be used later
#define max_error 0.05f

#define OFFSIDE 0
#define ONSIDE 1
#define POINT_STRIDE 3
struct point
{
	float data[dim_count];
};
struct heap_entry
{
	float value;
	unsigned char index;
};
struct stack_entry{
	size_t n;
	uint state;
};

__device__ float heapHeadValue(heap_entry* h)
{
	return h->value;
}

__device__ heap_entry* heapHeadReplace(heap_entry* h, const int index, const float value, const uint K)
{
	uint i = 0;
	for (; i < K - 1; ++i)
	{
		if (h[i + 1].value > value)
			h[i] = h[i + 1];
		else
			break;
	}
	h[i].value = value;
	h[i].index = index;
	return h;
}

__device__ heap_entry *heapInit(const uint K)
{
	heap_entry *h;
	for (uint i = 0; i < K; ++i)
		h[i].value = FLOAT_MAX;
	return h;
}
struct kd_node
{
	//Which dimension
	unsigned int dim;
	//At what value was this node split?
	int cutVal;
	//The index of the current node
	int index;
};


#define inx_size 12
struct /*__align__(inx_size)*/ indx
{
	//The points of the KD tree
	point *pts;
	//The linked nodes
	const kd_node *nodes;
};

//Just a utility function for converting an int equal to zero to one, and vice versa. Its not well optimized, but it was quick to write :P Would be better with bitshifts
__device__ int flip(int in)
{
	return abs(in - 1);
}
__device__ unsigned int childLeft(const unsigned int pos) { return 2*pos + 1; }
__device__ unsigned int childRight(const unsigned int pos) { return 2*pos + 2; }
struct maxAB
{
	float A,B;
	int indx_a, indx_b;
};

//Clamp the value to 1 or 0
__device__ static float intensity(float a)
{
	return fmax(1,fmin(0,fabs(a)));
}
struct heap
{
	heap_entry *entries;
	int current_count;
};
//If dynamic parrallelism is not available, default to compute model 3_2. Eg: The early 700 series
#ifndef CM3_5
#define CM3_2
#endif
//Used to see if we're within bounds and are ready to jump a node
__device__ unsigned int withinBounds(int cd, point q, point p, float heapHeadVal, float maxRad, float maxError)
{
	float diff = q.data[cd] -p.data[cd];
	float side2 = diff*diff;
	if ((side2 <= maxRad) &&
		(side2 * maxError < heapHeadVal))
	{ 
		return 1;
	}
	return 0;
}
//Used for warp devices. One if distance is greater than zero. Returns 0 or 1 
__device__ unsigned int nodeMinor(int cd, point q, point p)
{
	float diff = q.data[cd] -p.data[cd];
	return (unsigned int)intensity(diff); 
	
}
//Implementation details: http://on-demand.gputechconf.com/gtc/2012/presentations/S0079-Warped-Parallel-Nearest-Neighbor-Searches-Using-KD-Trees.pdf
__device__ void recursive_warp_search(const indx static_data, const point query_point,  unsigned int _Mask, heap *output, 
					uint stackpointer, stack_entry *stack, stack_entry *s)
{
	stackpointer--;
	const size_t n = s->n;
	const kd_node node = static_data.nodes[n];
	const int cd = node.cutVal;
	//Continue doesn't do anything anymore since we're in a __device__ function (Not __global__), and there is no while loop
	/*if (cd == -2)
		continue;*/
	const int index = node.index;
	point p = static_data.pts[index];
	// compute new distance and update if lower
	float dist = 0;
	for (uint i = 0; i < dim_count; ++i)
	{
		const float diff = query_point.data[i] - p.data[i];
		dist += diff * diff;
	}
	if ((dist <= max_rad) &&
		(dist < heapHeadValue(output->entries)) &&
		(dist > (float)max_error)){
		output->entries = heapHeadReplace(output->entries, index, dist, K_size);output->current_count++;}
		// look for recursion
	//Let the warp group decide which way we want to travel next
	_Mask = _Mask & __ballot(nodeMinor(cd, query_point,p));
	
	
	//If side >= 0, then we branch right first
	if(_Mask)
	{
		s->n = childRight(n);
		recursive_warp_search(static_data, query_point,  _Mask, output, 
					stackpointer, stack, s);
		stackpointer++;
		s = stack[stackpointer];
		//This needs to be called before the __any, since the thread needs to be terminated before we conduct the next vote.
		if(output->current_count > K_size)
		{	
			/*Exit the kernel if we have all of the points that we need. Since all of the points are clustered, hopefully this is greatly reduced and all threads exit
			at near the same time*/
			return;
		}
		//If any of the remaining active threads are within bounds of the left node
		if(__any(withinBounds(cd,query_point, p, heapHeadValue(ouput->entries), max_rad, max_error)))
		{
			s->n = childLeft(n);
			recursive_warp_search(static_data, query_point,  _Mask, output, 
				stackpointer, stack, s);
			stackpointer++;
		}
	}
	//Otherwise we branch left
	else
	{
		s->n = childLeft(n);
		recursive_warp_search(static_data, query_point,  _Mask, output, 
					stackpointer, stack, s);
		stackpointer++;
		s = stack[stackpointer];
		if(output->current_count > K_size)
		{	
			/*Exit the kernel if we have all of the points that we need. Since all of the points are clustered, hopefully this is greatly reduced and all threads exit
			at near the same time*/
			return;
		}
		//If any of the remaining active threads are within bounds of the right node
		if(__any(withinBounds(cd,query_point, p, heapHeadValue(ouput->entries), max_rad, max_error)))
		{
			s->n = childRight(n);
			recursive_warp_search(static_data, query_point,  _Mask, output, 
				stackpointer, stack, s);
			stackpointer++;
		}
	}
	//TODO: Sort
	
} 
/*Kernel is to be executed as 32x1
indx is pre malloced and copied to the GPU to avoid memory bottlenecks. Query points is copied per iteration.
Uses a warped ballot system. Preferable for clustered points that are closely together.
Make sure the thread group size is equal to the size of the cluster & is a multiple of 32*/
__global__ void clustered_search(indx static_data, const point *query_points, int *indices,  heap *ret, int query_amt)
{
	stack_entry stack[maximum_depth];
	//Global thread number
	int thread_num = blockIdx.x * BLOCK_SIZE + threadIdx.x;
	heap myHeap;
	myHeap.entries = heapInit(K_size);
	myHeap.current_count = 0;
	//Start at root node
	stack_entry* s = stack;
	uint startpos = 1;
	recursive_warp_search(static_data, query_points[thread_num], 1, &myHeap, startpos,s,stack);
	ret[thread_num] = myHeap;
} 

